#include "hip/hip_runtime.h"

extern "C" __global__ void simple(occaKernelInfoArg, int N, float *d_a){
	   
  // Convert thread and thread-block indices into array index 
  const int n  = threadIdx.x + blockDim.x*blockIdx.x;
	   
  // If index is in [0,N-1] add entries
  if(n<N)
    d_a[n] = n;
}
